#include "hip/hip_runtime.h"
#define NSPECIES 1
#define NPARAM 3
#define NREACT 2

//Code for texture memory
__device__ float function_1(float a1){
    return a1;
}


__device__ void step(float *y, float t, unsigned *rngRegs, int tid){

    float d_y0= DT * ((1.0*(tex2D(param_tex,0,tid)*function_1(tex2D(param_tex,1,tid)))-1.0*(tex2D(param_tex,0,tid)*tex2D(param_tex,2,tid)*y[0]))/tex2D(param_tex,0,tid));


    d_y0 += ((1.0*sqrt(tex2D(param_tex,0,tid)*function_1(tex2D(param_tex,1,tid)))*randNormal(rngRegs,sqrt(DT))-1.0*sqrt(tex2D(param_tex,0,tid)*tex2D(param_tex,2,tid)*y[0])*randNormal(rngRegs,sqrt(DT)))/tex2D(param_tex,0,tid));

    y[0] += d_y0;
}
//Code for shared memory
__device__ float function_1(float a1){
    return a1;
}


__device__ void step(float *parameter, float *y, float t, unsigned *rngRegs){

    float d_y0= DT * ((1.0*(parameter[0]*function_1(parameter[1]))-1.0*(parameter[0]*parameter[2]*y[0]))/parameter[0]);


    d_y0+= ((1.0*sqrt(parameter[0]*function_1(parameter[1]))*randNormal(rngRegs,sqrt(DT))-1.0*sqrt(parameter[0]*parameter[2]*y[0])*randNormal(rngRegs,sqrt(DT)))/parameter[0]);

    y[0] += d_y0;
}
