
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

#define NCOLS 163317 // number of columns
#define NROWS 8 // number of rows
#define nTPB 1024  // Threads per Block. nTPB should be a power-of-2
#define MAX_BLOCKS_X ((NCOLS/nTPB)+1) // # of blocks I will launch

#define FLOAT_MIN -1.0f // lowest anticipated number of the data. Values in array will be compared with this and updated with the larger one


__device__ volatile float blk_vals[NROWS][MAX_BLOCKS_X];
__device__ volatile int   blk_idxs[NROWS][MAX_BLOCKS_X];
// blk_vals and blk_idxs are the results obtained from reduction within each block.
// after 1st reduction, each row will have blk_vals[MAX_BLOCKS_X] array and blk_idxs[MAX_BLOCKS_X]
// these will be passed to the 2nd kernel

__global__ void max_idx_kernel_reduction_within_block(const float *data, const int xSize, const int ySize){  // first kernel. Reduction within blocks
  __shared__ volatile float vals[nTPB]; // Total amount of shared memory per block: 49152 bytes (50 KB). 1024 gives ~ 4KB for single.
  __shared__ volatile int idxs[nTPB]; // ~ 4 KB for single, when nTPB is 1024. each block will have both indices and values

  int idx = threadIdx.x+blockDim.x * blockIdx.x; // idx in the x direction
  int idy = blockIdx.y;
  float my_val = FLOAT_MIN; // lowest possible number
  int my_idx = -1;  // to check whether you DID perform the kernel. Again, it's the idx in the x dir.

  // sweep from global memory
  while (idx < xSize){   // this ensures you don't go out the size of the array's x direction
    float temp = data[idy*xSize+idx];
    if (temp > my_val) {my_val = temp; my_idx = idx;}
    // compare with my_val, and put the bigger value into my_val for next comparison. my_idx is 0 index based
    idx += blockDim.x*gridDim.x;}
                                                                 // until here takes about 6 ms !! very fast!!
  // populate shared memory: takes ~ 270 ms
  vals[threadIdx.x] = my_val;  // put the computed max value for each thread into the shared memory. -> this is the bottleneck!!
  idxs[threadIdx.x] = my_idx;  // do this for index as well -> this is also slow!!

  __syncthreads();

  // sweep in shared memory
  for (int i = (nTPB>>1); i > 0; i>>=1){
    if (threadIdx.x < i)    // the first half threads of the block
      if (vals[threadIdx.x] < vals[threadIdx.x + i]) {vals[threadIdx.x] = vals[threadIdx.x+i]; idxs[threadIdx.x] = idxs[threadIdx.x+i]; }
                            // the above is comparing shared memory of threadIdx.x with shared memory of threadIdx.x + i.
                            // then puts the larger value into shared memory of threadIdx.x
    __syncthreads();}       // so now in each block, shared memory's first element (index 0) is the max value and max value index


  // perform block-level reduction
  if (!threadIdx.x){    // at the shared memory, only the first element (index 0) (actually 2 elements in the first index. max value, and max value index) is what we need.
      blk_vals[blockIdx.y][blockIdx.x] = vals[0]; // For each window (single x row), the first elements of the blocks are stored into the blk_vals[windowNumber][:]
                                                // remember, this is a global variable.
      blk_idxs[blockIdx.y][blockIdx.x] = idxs[0]; // and the max value index

  __syncthreads();
}

}

  // originally the following kernel was in the 1st kernel, performed by the last block. So just use one block for this.
__global__ void max_idx_kernel_final(int *result_maxInd, float *result_maxVal){

  __shared__ volatile float vals[nTPB]; //  Total amount of shared memory per block: 49152 bytes (50 KB). 1024 gives ~ 4KB for single.
  __shared__ volatile int idxs[nTPB]; // ~ 4 KB for single, when nTPB is 1024. each block will have these variables!! (vals and idxs)

  int idx = threadIdx.x;
  int idy = blockIdx.y;
  float my_val = FLOAT_MIN;
  int my_idx = -1;  // remember, these are local variables, so each thread has this variable. This local variable is independent from other thread's local variable
  while (idx < MAX_BLOCKS_X ){                                                          // ?? confused whether it should be gridDim.x (actual # of blocks launched) or MAX_BLOCKS_X (# of elements in x dir of the global array blk_vals)
    float temp = blk_vals[idy][idx];
    if (temp > my_val)
        {my_val = temp; my_idx = blk_idxs[idy][idx]; }
    idx += blockDim.x;} // all threads in this single block (single in the x dir) are working, so you should loop over blockDim.x.
                      // Imagine where gridDim.x (# of blocks) is huge so that you need to loop over to get the max value and index
                      // After this, each thread in the block has a local variable (max value and max value index).
                      // So far it was sort of a reduction, but instead of pairing values we just looped over the blk_vals and blk_idxs
  // populate shared memory
  idx = threadIdx.x;
  vals[idx] = my_val;   // This is now shared memory. This is because reduction requires comparison between different elements
  idxs[idx] = my_idx;   // my_idx value is 0 based. This is done for all blocks (in the y direction)
  __syncthreads();
  // Now the final task is to do reduction for all threads in our single block (single block in the x dir, NROWS blocks in the y dir)!

// sweep in shared memory
  for (int i = (nTPB>>1); i > 0; i>>=1) {
    if (idx < i) // the first half threads of the block
      if (vals[idx] < vals[idx + i]) {vals[idx] = vals[idx+i]; idxs[idx] = idxs[idx+i]; }
    __syncthreads();} // now all the results are in threadIdx.x == 0 for each block (there are NROWS blocks in the y dir)
  // 0th thread. the results are in shared memory, not the local memory, so any thread could do the following. We just selected the 0th thread for no reason. If several threads try to do this, that would be a problem, since we'll have to wait for them

  if(!threadIdx.x){
        result_maxInd[idy] = idxs[0]; // the final result for each row goes into the corresponding position (blockIdx.y)
        result_maxVal[idy] = vals[0];
      }
}


int main(){

    dim3 grids(MAX_BLOCKS_X, NROWS); //(160,8,1)
    dim3 threads(nTPB,1); //(1024,1,1)
    dim3 grids2(1,NROWS); //(1,8,1)
    dim3 threads2(nTPB); //(1024,1,1)

    float *d_vector, *h_vector;

    h_vector = (float*)malloc(NROWS * NCOLS * sizeof(float));
    memset(h_vector, 0, NROWS*NCOLS*sizeof(float));

    for (int i =  0; i < NROWS; i++){
      h_vector[i*NCOLS + i] = 10.0f;  // create definite max element per row
      printf("%f\n", h_vector[i*NCOLS + i]);
    }
    hipMalloc(&d_vector, NROWS * NCOLS * sizeof(float));
    hipMemcpy(d_vector, h_vector, NROWS * NCOLS * sizeof(float), hipMemcpyHostToDevice);

    //d_vector is a pointer on the device pointing to the beginning of the vector, containing nrElements floats.

    int *max_index;
    float *max_val;
    int *d_max_index;
    float *d_max_val;

    max_index = (int*)malloc(NROWS * sizeof(int));
    max_val = (float*)malloc(NROWS * sizeof(float));
    hipMalloc((void**)&d_max_index, NROWS * sizeof(int));
    hipMalloc((void**)&d_max_val, NROWS * sizeof(float));

    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);
    max_idx_kernel_reduction_within_block<<<grids, threads>>>(d_vector, NCOLS, NROWS);
    max_idx_kernel_final<<<grids2,threads2>>>(d_max_index, d_max_val);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float et;
    hipEventElapsedTime(&et, start, stop);
    printf("elapsed time: %fms\n", et);

    hipMemcpy(max_index, d_max_index, NROWS * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(max_val, d_max_val, NROWS * sizeof(float), hipMemcpyDeviceToHost);

    for(int z=0;z<NROWS;z++)
      printf("%d  ",max_index[z]);

    printf("\n\n\n");

    for(int z=0;z<NROWS;z++)
      printf("%f  ",max_val[z]);
    printf("\n");
    return 0;
}
